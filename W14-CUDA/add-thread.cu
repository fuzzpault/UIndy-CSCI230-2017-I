#include "hip/hip_runtime.h"
/*                                                                                                                    
  Name: Paul Talaga
  Date: Dec 1, 2017
  Desc: Program to add two arrays using the GPU
        Multiple threads but 1 block, so this will NOT be superfast, 
        but it is using the GPU.

  To compile this, do: nvcc add-thread.cu 
*/

#include <iostream>

using namespace std;

#define numThreads 32 // Max is 1024


// CUDA kernel function to add to arrays element by element
// This will add some of the elements in the array, but over
// all threads the entire array will be computed.
__global__
void add(int size, int* x, int* y, int* z){
    //threadIdx.x
    //blockIdx.x
    int index = threadIdx.x;
    int stride = gridDim.x; // gridDim is numThreada
    for(int i = index; i < size; i+= stide){
        z[i] = x[i] + y[i];
    }
}

int main(void){
    
    // Size of the arrays we'll be adding
    const unsigned N = 10000000;
    
    cout << "Size: " << N*3*sizeof(int) / 1024 / 1024 << " MB" << endl;
    
    // To used unified memory (CUDA takes care of data movement)
    // all memory must be allocated via the hipMallocManaged call below.
    int* x;
    int* y;
    int* z;
    
    hipMallocManaged(&x, N * sizeof(int));
    hipMallocManaged(&y, N * sizeof(int));
    hipMallocManaged(&z, N * sizeof(int));
    
    // Fill the arrays with numbers
    for(int i = 0; i < N; i++){
        x[i] = i;
        y[i] = 2 * i;
    }
    
    // Call the add function, with 1 block, and 1 thread
    add<<<numBlocks,numThreads>>>(N, x, y, z);
    
    // Wait until the device is done before proceeding, otherwise we'd be
    // accessing x, y, and z in the loop below before the add function completes
    // on the device.
    hipDeviceSynchronize();
    
    // Check to see if the math is correct
    int errors = 0;
    for(int i = 0; i < N; i++){
        if(z[i] != x[i] + y[i]){
            cout << i << " did not add correctly!" << endl;
            errors++;
        }
    }
    if(!errors)cout << "All good!" << endl;
    
    return 0;
}