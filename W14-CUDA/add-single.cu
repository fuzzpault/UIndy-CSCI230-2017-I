/*                                                                                                                    
  Name: Paul Talaga
  Date: Nov 27, 2017
  Desc: Program to add two arrays using the GPU
        Only 1 thread in 1 block, so this will NOT be fast, 
        but it is using the GPU

  To compile this, do: nvcc add-single.cu 
*/


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// CUDA kernel function to add to arrays element by element
// This will add all elements in the array in 1 call.
__global__
void add(int size, int* x, int* y, int* z){
    for(int i = 0; i < size; i++){
        z[i] = x[i] + y[i];
    }
}

int main(){
    
    // Size of the arrays we'll be adding
    const unsigned N = 100;
    
    // To used unified memory (CUDA takes care of data movement)
    // all memory must be allocated via the cudaMallocManaged call below.
    int* x;
    int* y;
    int* z;
    
    hipMallocManaged(&x, N * sizeof(int));
    hipMallocManaged(&y, N * sizeof(int));
    hipMallocManaged(&z, N * sizeof(int));
    
    // Fill the arrays with numbers
    for(int i = 0; i < N; i++){
        x[i] = i;
        y[i] = 2 * i;
    }
    
    // Call the add function, with 1 block, and 1 thread
    add<<<1,1>>>(N, x, y, z);
    
    // Wait until the device is done before proceeding, otherwise we'd be
    // accessing x, y, and z in the loop below before the add function completes
    // on the device.
    hipDeviceSynchronize();
    
    // Check to see if the math is correct
    int errors = 0;
    for(int i = 0; i < N; i++){
        if(z[i] != x[i] + y[i]){
            cout << i << " did not add correctly!" << endl;
            errors++;
        }
    }
    if(!errors)cout << "All good!" << endl;
    
    return 0;
}