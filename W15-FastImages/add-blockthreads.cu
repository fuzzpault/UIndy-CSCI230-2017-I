/*                                                                                                                    
  Name: Paul Talaga
  Date: Dec 5, 2017
  Desc: Program to add two arrays using the GPU
        It uses multiple threads and multiple blocks, so this is
        as fast it can go.

  To compile this, do: nvcc add-blockthreads.cu 
*/


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define numThreads 32 // upper limit of 1024
#define numBlocks 1000


// CUDA kernel function to add to arrays element by element
// This isn't a great demonstration because there isn't much
// work for this function to do, so you'll likely not see
// a big speedup over the CPU.
__global__
void add(int size, int* x, int* y, int* z){
    //threadIdx.x
    //blockIdx.x
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // gridDim is numThreads
    for(int i = index; i < size; i+= stride){
        z[i] = x[i] + y[i];
    }
}

int main(void){
    
    // Size of the arrays we'll be adding
    const unsigned N = 10000000;
    
    cout << "Size: " << N*3*sizeof(int) / 1024 / 1024 << " MB" << endl;
    
    // To used unified memory (CUDA takes care of data movement)
    // all memory must be allocated via the cudaMallocManaged call below.
    int* x;
    int* y;
    int* z;
    
    hipMallocManaged(&x, N * sizeof(int));
    hipMallocManaged(&y, N * sizeof(int));
    hipMallocManaged(&z, N * sizeof(int));
    
    // Fill the arrays with numbers
    for(int i = 0; i < N; i++){
        x[i] = i;
        y[i] = 2 * i;
    }
    
    // Call the add function, with 1 block, and 1 thread
    add<<<numBlocks,numThreads>>>(N, x, y, z);
    
    // Wait until the device is done before proceeding, otherwise we'd be
    // accessing x, y, and z in the loop below before the add function completes
    // on the device.
    hipDeviceSynchronize();
    
    // Check to see if the math is correct
    int errors = 0;
    for(int i = 0; i < N; i++){
        if(z[i] != x[i] + y[i]){
            cout << i << " did not add correctly!" << endl;
            errors++;
        }
    }
    if(!errors)cout << "All good!" << endl;
    
    return 0;
}